
#include <hip/hip_runtime.h>
#include <iostream>
#include <iterator>
#include <vector>
#include <algorithm>
#include <cassert>
#include <cstdlib>
using namespace std;

void checkResult(int *vec1, int *vec2, int *resultVec, int size) {
  for (int i = 0; i < size; i++) {
    assert(resultVec[i] == vec1[i] + vec2[i]);
  }
}




// CUDA kernel for vector addition
__global__ void addVectors(int* vec1, int* vec2, int* resultVec, int size) {
  int threadId = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (threadId < size) {
    resultVec[threadId] = vec1[threadId] + vec2[threadId];
  }
}



int main() {
  constexpr int arraySize = 1 << 26;
  size_t memorySize = sizeof(int) * arraySize;

  int *hostVec1, *hostVec2, *hostResult;
  hipHostMalloc(&hostVec1, memorySize, hipHostMallocDefault);
  hipHostMalloc(&hostVec2, memorySize, hipHostMallocDefault);
  hipHostMalloc(&hostResult, memorySize, hipHostMallocDefault);

  
  for(int i = 0; i < arraySize; i++){
    hostVec1[i] = rand() % 100;
    hostVec2[i] = rand() % 100;
  }



  
  int *devVec1, *devVec2, *devResult;
  hipMalloc(&devVec1, memorySize);
  hipMalloc(&devVec2, memorySize);
  hipMalloc(&devResult, memorySize);

  hipMemcpy(devVec1, hostVec1, memorySize, hipMemcpyHostToDevice);
  hipMemcpy(devVec2, hostVec2, memorySize, hipMemcpyHostToDevice);



  
  int threadsPerBlock = 1 << 10;
  int blocksPerGrid = (arraySize + threadsPerBlock - 1) / threadsPerBlock;

  
  addVectors<<<blocksPerGrid, threadsPerBlock>>>(devVec1, devVec2, devResult, arraySize);
  hipMemcpy(hostResult, devResult, memorySize, hipMemcpyDeviceToHost);

  
  checkResult(hostVec1, hostVec2, hostResult, arraySize);


  
  hipHostFree(hostVec1);
  hipHostFree(hostVec2);
  hipHostFree(hostResult);
  hipFree(devVec1);
  hipFree(devVec2);
  hipFree(devResult);


  
  cout << "COMPLETED SUCCESSFULLY\n";

  return 0;
}
